#include "hip/hip_runtime.h"
#include "../planner.hpp"
#include "../multi_dubins/multi_dubins.hpp"
#include "hip/hip_runtime.h"

struct point_t{
    double x,y;
};

__global__ void get_safe_curve_cuda(
    double* x_components,
    double* y_components, 
    point_t* new_a_arr,
    dubins::d_curve* out_arr,
    const double r,
    const int N_points
){

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= N_points-2)
        return;

    point_t a = {x_components[id], y_components[id]};
    point_t b = {x_components[id + 1], y_components[id + 1]};
    point_t c = {x_components[id + 2], y_components[id + 2]};

    double vx0 = b.x - a.x;
    double vy0 = b.y - a.y;
    double th0 = atan2(vy0, vx0); // * sgn(vy0);

    double norm0 = sqrt(vx0 * vx0 + vy0 * vy0);
    double unitx0 = vx0 / norm0;
    double unity0 = vy0 / norm0;

    double vxf = c.x - b.x;
    double vyf = c.y - b.y;
    double thf = atan2(vyf, vxf); // * sgn(vyf);

    double normf = sqrt(vxf * vxf + vyf * vyf);
    double unitxf = vxf / normf;
    double unityf = vyf / normf;

    /*
    * |A·B| = |A| |B| cos(θ)
    * |A×B| = |A| |B| sin(θ)
    * with this we can easily get the angle between the two vectors
    * we add fabs to normalize in [0, pi)
    */
    
    //old version with sincos
/*   
    double cross_prod_3_component = vx0 * vyf - vy0 * vxf;
    double alpha = M_PI - atan2(fabs(cross_prod_3_component), vx0 * vxf + vy0 * vyf); //angle between vectors
    double sina, cosa;
    sincos(alpha / 2., &sina, &cosa);
    double d = r * (cosa / sina);
    double xf = b.x + d * unitxf;
    double yf = b.y + d * unityf;
    new_a_arr[id] = {xf, yf};
*/
    

    double cross_prod_3_component = vx0 * vyf - vy0 * vxf;
    double abs_cross_prod = fabs(cross_prod_3_component);
    double alpha = M_PI - atan2(abs_cross_prod, vx0 * vxf + vy0 * vyf); //angle between vectors
    double d = r * (abs_cross_prod / (vx0 * vxf + vy0 * vyf + normf * norm0));
    double xf = b.x + d * unitxf;
    double yf = b.y + d * unityf;
    new_a_arr[id] = {xf, yf};

    point_t turning_point{b.x + d * -unitx0, b.y + d * -unity0};
    double straight_segment_len = sqrt((turning_point.x - a.x)*(turning_point.x - a.x) + (turning_point.y - a.y)*(turning_point.y - a.y));

    dubins::d_curve curve = {
        .a1 = {a.x, a.y, th0, 0, 0, a.x, a.y, th0}, // garbage
        .a2 = {a.x, a.y, th0, 0, straight_segment_len, turning_point.x, turning_point.y, th0},
        .a3 = {turning_point.x, turning_point.y, th0, ((cross_prod_3_component > 0) - (cross_prod_3_component < 0)) / r, (M_PI - alpha) * r, xf, yf, thf},
        .L = straight_segment_len + (M_PI - alpha) * r
    };

    out_arr[id] = curve;

    __syncthreads();

    if(id != N_points - 3){
        out_arr[id + 1].a2.x0 = xf;
        out_arr[id + 1].a2.y0 = yf;
        out_arr[id + 1].a2.L = sqrt((xf - out_arr[id + 1].a2.xf) * (xf - out_arr[id + 1].a2.xf) + 
            (yf - out_arr[id + 1].a2.yf) * (yf - out_arr[id + 1].a2.yf));    
    }
}

void Planner::dubins_wrapper(const VisiLibity::Polyline& path, multi_dubins::path_t& sol, VisiLibity::Point& new_a, double r){
    std::cerr << "\n----CUDA ALGO----\n";

    std::vector<double> x_components_h, y_components_h;
    std::vector<point_t> new_a_arr_h{ sol.size() - 2 }; // first and last curves dont generate a new_a point
    int threads = 32;
    int blocks = (sol.size() + threads - 1) / threads; 

    for (uint64_t i = 1; i < path.size(); ++i){
        x_components_h.push_back(path[i].x());
        y_components_h.push_back(path[i].y());
    } 
    
    int n_bytes_x_components = sizeof(double) * x_components_h.size();
    int n_bytes_out_arr = sizeof(dubins::d_curve) * new_a_arr_h.size(); // n. of curves = n. of new_a points
    int n_bytes_new_a_arr = sizeof(point_t) * new_a_arr_h.size();

    double *x_components, *y_components;
    hipMalloc(&x_components, n_bytes_x_components);
    hipMalloc(&y_components, n_bytes_x_components);

    point_t *new_a_arr_dev;
    hipMalloc(&new_a_arr_dev, n_bytes_new_a_arr);

    dubins::d_curve* out_arr_dev;
    hipMalloc(&out_arr_dev, n_bytes_out_arr);

    hipMemcpy(x_components, x_components_h.data(), n_bytes_x_components, hipMemcpyHostToDevice);
    hipMemcpy(y_components, y_components_h.data(), n_bytes_x_components, hipMemcpyHostToDevice);

    auto start_time = std::chrono::system_clock::now();
    get_safe_curve_cuda<<<blocks, threads>>>(x_components, y_components, new_a_arr_dev, out_arr_dev, r, x_components_h.size());
    auto end_time = std::chrono::system_clock::now();
    std::cout << "time elapsed: " << std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() << " us\n";
    //std::cout << hipDeviceSynchronize()<<"\n";

    hipMemcpy(sol.data()+1, out_arr_dev, n_bytes_out_arr, hipMemcpyDeviceToHost); // +1 leaves space for 1° curve
    hipMemcpy(new_a_arr_h.data(), new_a_arr_dev, n_bytes_new_a_arr, hipMemcpyDeviceToHost);

    /*
    for(int i = 2; i < sol.size()-1; ++i){
        sol[i].a2.x0 = new_a_arr_h[i-2].x;
        sol[i].a2.y0 = new_a_arr_h[i-2].y;
        sol[i].a2.L = sqrt(((sol[i].a2.xf - sol[i].a2.x0)*(sol[i].a2.xf - sol[i].a2.x0) + 
            (sol[i].a2.yf - sol[i].a2.y0)*(sol[i].a2.yf - sol[i].a2.y0)));
        
    }
    sol[1].a2.L = sqrt(((sol[1].a2.xf - sol[1].a2.x0)*(sol[1].a2.xf - sol[1].a2.x0) + 
            (sol[1].a2.yf - sol[1].a2.y0)*(sol[1].a2.yf - sol[1].a2.y0)));
    */

    new_a = {new_a_arr_h.back().x, new_a_arr_h.back().y};

}
