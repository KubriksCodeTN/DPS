#include "hip/hip_runtime.h"
/**
 * @file dubins_cuda.cu
 * @brief contains the test functions for the parallel DPS implementation
 */
#include "hip/hip_runtime.h"
#include "planner.hpp"
#include <iostream>
#include <chrono>
#include <stdio.h>
#include <cassert>

struct point_t{
    double x, y;
};

/*
struct lk_word{
    point_t p0, q0, q1;
    double th0, thf;
    double k;
};
*/

struct lk_word{
    point_t *p0, *q0, *q1;
    double *th0, *thf;
    double *k;
};

__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}

/**
 * @brief calculates the solution using DPS interpolation, the solution pieces are computed in parellel on the GPU
 * 
 * @param x_components x components of the polyline
 * @param y_components y components of the polyline
 * @param [out] p0 array of starting points of the solution pieces
 * @param [out] q0 array of starting points of the arc in the solution pieces
 * @param [out] q1 array of ending points of the solution pieces
 * @param [out] th00 array of starting angles of the solution pieces
 * @param [out] thff array of ending angles of the solution pieces
 * @param [out] k auxiliary array to know the orientation of the arc in the solution piece
 * @param r curvature radius of the robot
 * @param N_curves number of solution pieces
 * 
 * @note some of these informations are redundant and could be removed
 */
__global__ void get_safe_curve_cuda(
    double* x_components,
    double* y_components, 
    point_t* p0,
    point_t* q0,
    point_t* q1,
    double* th00,
    double* thff,
    double* k,
    const double r,
    const int N_curves
){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= N_curves) // each thread creates a curve from 3 points a, b, c. the last straigth segment is not created here
        return;

    point_t a = {x_components[id], y_components[id]};
    point_t b = {x_components[id + 1], y_components[id + 1]};
    point_t c = {x_components[id + 2], y_components[id + 2]};

    double vx0 = b.x - a.x;
    double vy0 = b.y - a.y;
    double th0 = atan2(vy0, vx0); // * sgn(vy0);

    double norm0 = sqrt(vx0 * vx0 + vy0 * vy0);
    double unitx0 = vx0 / norm0;
    double unity0 = vy0 / norm0;

    double vxf = c.x - b.x;
    double vyf = c.y - b.y;
    double thf = atan2(vyf, vxf); // * sgn(vyf);

    double normf = sqrt(vxf * vxf + vyf * vyf);
    double unitxf = vxf / normf;
    double unityf = vyf / normf;

    /*
    * |A·B| = |A| |B| cos(θ)
    * |A×B| = |A| |B| sin(θ)
    * with this we can easily get the angle between the two vectors
    * we add fabs to normalize in [0, pi)
    */

    double cross_prod_3_component = vx0 * vyf - vy0 * vxf;
    double abs_cross_prod = std::fabs(cross_prod_3_component);
    double d = r * (abs_cross_prod / (vx0 * vxf + vy0 * vyf + normf * norm0));
    double xf = b.x + d * unitxf;
    double yf = b.y + d * unityf;
    double xq0 = b.x - d * unitx0;
    double yq0 = b.y - d * unity0;

    q0[id] = {xq0, yq0};    
    q1[id] = {xf, yf};
    th00[id] = th0;
    thff[id] = thf;
    k[id] = ((cross_prod_3_component > 0) - (cross_prod_3_component < 0)) / r;
    p0[id + 1] = {xf, yf};        
}

/**
 * @brief helper function to setup the cuda test
 */
double Planner::dubins_wrapper(const VisiLibity::Polyline &path_poly, multi_dubins::path_t &sol, VisiLibity::Point &new_a, double){
    // cuda startup
    hipFree(0);
    warm_up_gpu<<<1024, 512>>>();

    int32_t n = path_poly.size();

    double threads = 512; // check
    double blocks = (n + threads - 1) / threads;

    double* x;
    double* y;
    double* x_cuda;
    double* y_cuda;
    lk_word path;
    lk_word path_cuda;       

    hipHostMalloc(&x, n * sizeof(double));
    hipHostMalloc(&y, n * sizeof(double));
    hipHostMalloc(&(path.p0), n * sizeof(point_t));
    hipHostMalloc(&(path.q0), n * sizeof(point_t));
    hipHostMalloc(&(path.q1), n * sizeof(point_t));
    hipHostMalloc(&(path.th0), n * sizeof(double));
    hipHostMalloc(&(path.thf), n * sizeof(double));
    hipHostMalloc(&(path.k), n * sizeof(double));
    hipMalloc(&(path_cuda.p0), n * sizeof(point_t));
    hipMalloc(&(path_cuda.q0), n * sizeof(point_t));
    hipMalloc(&(path_cuda.q1), n * sizeof(point_t));
    hipMalloc(&(path_cuda.th0), n * sizeof(double));
    hipMalloc(&(path_cuda.thf), n * sizeof(double));
    hipMalloc(&(path_cuda.k), n * sizeof(double));

    for (auto i = 0; i < n; ++i){
        x[i] = path_poly[i].x();
        y[i] = path_poly[i].y();
        //printf("%lf, %lf\n", x[i], y[i]);
    }

    auto start = std::chrono::high_resolution_clock::now();
    hipMalloc(&x_cuda, n * sizeof(double));  
    hipMalloc(&y_cuda, n * sizeof(double));
    hipMemcpy(x_cuda, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y_cuda, y, n * sizeof(double), hipMemcpyHostToDevice);
    get_safe_curve_cuda<<<blocks, threads>>>(x_cuda, y_cuda, path_cuda.p0, path_cuda.q0, path_cuda.q1, path_cuda.th0, path_cuda.thf, path_cuda.k,.5, n - 2);   
    hipMemcpy(path.p0, path_cuda.p0, (n - 2) * sizeof(point_t), hipMemcpyDeviceToHost);
    hipMemcpy(path.q0, path_cuda.q0, (n - 2) * sizeof(point_t), hipMemcpyDeviceToHost);
    hipMemcpy(path.q1, path_cuda.q1, (n - 2) * sizeof(point_t), hipMemcpyDeviceToHost);
    hipMemcpy(path.th0, path_cuda.th0, (n - 2) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(path.thf, path_cuda.thf, (n - 2) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(path.k, path_cuda.k, (n - 2) * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    path.p0[0] = {x[0], y[0]};

    auto dist = [](const point_t& a, const point_t& b){
        return sqrt((a.x - b.x)  * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
    };

    // for(int i = 0; i<n-2; ++i){
    //     printf("(%lf,%lf)\n(%lf,%lf)\n(%lf,%lf)\n", path.p0[i].x, path.p0[i].y, path.q0[i].x,path.q0[i].y,path.q1[i].x,path.q1[i].y);
    // }

    //----- just for printing on desmos

    
    for (int i = 0; i < n - 2; ++i){
        double l2 = dist({path.p0[i].x, path.p0[i].y}, {path.q0[i].x, path.q0[i].y});
        double l3 = dist({path.q0[i].x, path.q0[i].y}, {path.q1[i].x, path.q1[i].y});
        l3 = 2 * inv_k * asin(l3 / 2 / inv_k); // formula for arc len given radius and coord of points
        sol[i] = {
            .a1 = {path.p0[i].x, path.p0[i].y, path.th0[i], 0, 0, path.p0[i].x, path.p0[i].y, path.th0[i]},
            .a2 = {path.p0[i].x, path.p0[i].y, path.th0[i], 0, l2, path.q0[i].x, path.q0[i].y, path.th0[i]},
            .a3 = {path.q0[i].x, path.q0[i].y, path.th0[i], path.k[i], l3, path.q1[i].x, path.q1[i].y, path.thf[i]},
            .L = l2 + l3
        };
    }

    // build the last straight curve
    point_t last_p0 = {sol[sol.size() - 2].a3.xf, sol[sol.size() - 2].a3.yf};
    double last_th0 = sol[sol.size() - 2].a3.thf;

    point_t last_q0 = {path_poly[path_poly.size() - 1].x(), path_poly[path_poly.size() - 1].y()};
    double last_len = dist(last_p0, last_q0);

    sol.back() = {
        .a1 = {last_p0.x, last_p0.y, last_th0, 0, 0, last_p0.x, last_p0.y, last_th0},
        .a2 = {last_p0.x, last_p0.y, last_th0, 0, last_len, last_q0.x, last_q0.y, last_th0},
        .a3 = {last_q0.x, last_q0.y, last_th0, 0, 0, last_q0.x, last_q0.y, last_th0},
        .L = last_len
    };


    return std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
}